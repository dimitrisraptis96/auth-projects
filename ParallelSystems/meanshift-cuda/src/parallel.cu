#include "hip/hip_runtime.h"
#include "../include/parallel.h"
#include "../include/global_vars.h"
#include "../include/cuda_helpers.h"

// constant device N and D
__device__ __constant__ int N_SIZE; 
__device__ __constant__ int D_SIZE;

// grid and block sizes
const int threads_per_block = 256;
const int blocks_per_grid = 32; 

typedef struct {
    int xid;
    double distance;
} SparseData;

// host copies
double *x_data, *y_data, *buffer, **x, **y;
int *nNbr;
SparseData **w; 
SparseData **tmpW;

// device copies
double *d_x_data,*d_y_data,*d_y_new_data,*d_m_data,*d_sum, *d_buffer;
double **d_x,**d_y,**d_y_new,**d_m;
int *d_nNbr;
SparseData **d_w,*d_sparse; 


__global__ void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData *w);
__global__ void gpu_normalize(int *nNbr, SparseData *w, double *y_new);

extern "C"
void parallel(){
  printf("[INFO]: CUDA-GPU IMPLEMENTATION\n");
  printf("=============================\n");

  struct timeval startwtime, endwtime;
  double seq_time;

  init_parallel();
  
  gettimeofday (&startwtime, NULL);
  //------------------------------
  meanshift();
  //------------------------------
  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

  /*printf("\n\nIs test PASSed? %s\n\n", validate_parallel()?"YES":"NO");
  printf("===============================================\n\n");*/
  printf("\n\n[INFO]: Parallel meanshift wall clock time = %f\n", seq_time);

}

void init_parallel(){
  // blocks_per_grid = MIN(32, (N+threads_per_block-1) / threads_per_block);

  cpu_malloc();
  gpu_malloc();
  read_file();
  move_data_to_gpu();
}

//Contiguous memory allocation for 2D arrays
void cpu_malloc(){
  int i;
  
  if(VERBOSE) printf("[INFO]: Allocating cpu memory..\n");

  // malloc pointers to rows 
  HANDLE_NULL( (x = (double **) malloc(N * sizeof(double *))) );
  HANDLE_NULL( (y = (double **) malloc(N * sizeof(double *))) );
  HANDLE_NULL( (w = (SparseData **) malloc(N * sizeof(SparseData *))) );

  // malloc data of the arrays
  HANDLE_NULL( (x_data = (double *) malloc(N * D * sizeof(double))) );
  HANDLE_NULL( (y_data = (double *) malloc(N * D * sizeof(double))) );
  HANDLE_NULL( (nNbr = (int *) malloc(N * sizeof(int))) );

  // assign pointers of data to arrays
  for(i=0; i < N; i++){
    x[i]      = x_data      + i * D;
    y[i]      = y_data      + i * D;
  }
}


// Allocate memory for devive variables and copy x host data to device
void gpu_malloc (){
  int size; 

  if(VERBOSE) printf("[INFO]: Allocating device memory..\n");
  
  // malloc pointers of rows
  size = N * sizeof(double *);
  HANDLE_ERROR( hipMalloc((void**)&d_x,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m,    size) );


  // malloc data of the arrays
  size = N * D * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_x_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new_data,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m_data,    size) );

  // malloc d_sum
  size = blocks_per_grid * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_sum, size) );

  // malloc d_nNbr
  size = N * sizeof(int);
  HANDLE_ERROR( hipMalloc((void**)&d_nNbr, size) );

  // malloc d_buffer
  size = N * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_buffer, size) );

  // malloc d_w indexes of rows
  size = N * sizeof(SparseData *);
  HANDLE_ERROR( hipMalloc((void**)&d_w, size) );

    // get back indexes from device (need them in rangesearch2sparse)
  // HANDLE_ERROR( hipMemcpy(tmpW, d_w, sizeof(SparseData**), hipMemcpyDeviceToHost) );
}


void move_data_to_gpu(){
  if(VERBOSE) printf("[INFO]: Move data to device..\n");

  // move device constant variables N_SIZE and D_SIZE
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(N_SIZE), &N, sizeof(int)) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE), &D, sizeof(int)) );

  // move device global variable d_x and d_x_data
  HANDLE_ERROR( hipMemcpy(d_x,      x,      N*sizeof(double *), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_x_data, x_data, N*D*sizeof(double), hipMemcpyHostToDevice) );
}

void free_memory(){
  int i;
  if (VERBOSE) printf("[INFO]: Deallocating memory...\n");
  //free() data
  for (i=0; i<N; i++){
    free(x[i]);
    free(y[i]);
  }
  // free() pointers
  free(x);
  free(y);
}

void read_file(){
  int i,j;

  FILE * fp;
  fp = fopen (DATASET_PATH, "r");

  if (fp == NULL) { perror("[ERROR]: "); exit(1); }

  for (i=0; i<N; i++) 
    for (j=0; j<D; j++)
      if (EOF ==  fscanf(fp, "%lf", &x[i][j])) { perror("[ERROR]:"); exit(1); }

  fclose(fp);
}

void write_csv_file (char *message, double **a, const int ROW, const int COL){
  int i,j;

  FILE * fp;
  fp = fopen (OUTPUT_PATH_PARALLEL, "w");

  if (fp == NULL){ perror("[ERROR]: "); exit(1); }

  fprintf(fp,"%s",message);

  for (i=0; i<ROW; i++) {
    for (j=0; j<COL; j++)
      if (EOF ==  fprintf(fp, "%lf, ", a[i][j])) {
        perror("[ERROR]:"); exit(1);
      }
    fprintf(fp,"\n");
  }

  fclose(fp);
}


void meanshift(){
  clock_t start, end;
  double cpu_time_used;
  
  int iter=0;
  double norm = DBL_MAX;

  gpu_init_arr <<<blocks_per_grid, threads_per_block>>> (d_nNbr, d_x_data, d_y_data, d_m_data);

  
  while (norm > EPSILON){
    iter++;
    
    //======================================================
    start = clock();

    // find distances and calculate kernels
    rangesearch2sparse();

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("rangesearch2sparse: %f\n", cpu_time_used);
    //======================================================

    //======================================================
    start = clock();
    // compute new y vector
    gpu_matrix_mult <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_x_data,d_y_new_data,d_sparse);

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("mult: %f\n", cpu_time_used);
    //======================================================

    //======================================================
    start = clock();

    // normalize vector
    gpu_normalize <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_sparse,d_y_new_data);    

    // calculate meanshift
    gpu_calc_meanshift <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_y_new_data,d_y_data);

    // update y
    gpu_copy_2Darray <<<blocks_per_grid, threads_per_block>>>(d_y_new_data, d_y_data);

    // calculate Frobenius norm
    gpu_frob_norm_shared <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_sum);
    norm = sqrt ( finish_reduction() );

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("other: %f\n", cpu_time_used);
    //======================================================

    if (VERBOSE) printf("[INFO]: Iteration %d - error %lf\n", iter, norm);   
  }

  // copy results back to host
  if (VERBOSE){  
    HANDLE_ERROR( hipMemcpy(y_data, d_y_data, N*D*sizeof(double), hipMemcpyDeviceToHost) );
    write_csv_file("",y,N,D);
  }

  // gpu_free_memory();
  // cpu_free_memory();
}

// TODO: shared memory: the data within the block
__global__
void gpu_init_arr(int *nNbr, double *x, double *y, double *m)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;

  while (tid < N_SIZE*D_SIZE) {
    // nNbr[tid%N_SIZE] = 0;
    y[tid] = x[tid];
    m[tid] = DBL_MAX;
    
    tid += blockDim.x * gridDim.x;
  }

}
// TODO: reduction with shared memory
__global__ void gpu_calc_distances
(int y_row, double h, double *buffer, double *y, double *x, double *n){

  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, x_arr_offset, y_arr_offset;
  
  double dist;

  while (tid < N_SIZE) {
    // diagonal elements
    if (y_row == tid){
      buffer[tid] = 1;
      tid += blockDim.x * gridDim.x;
      continue;
    }
    
    x_arr_offset = tid*D_SIZE;
    y_arr_offset = y_row*D_SIZE;  

    dist = 0; 
    for(k=0; k<D_SIZE; k++){
      dist += (y[y_arr_offset + k] - x[x_arr_offset + k])*(y[y_arr_offset + k] - x[x_arr_offset + k]);
    }

    // element inside radious
    if (dist < h*h){
      buffer[tid]= exp(-dist / (2.0*h*h));
    }
    // unnecessary elements
    else{
      buffer[tid]=0;
    }

    tid += blockDim.x * gridDim.x;
  }
}

void rangesearch2sparse(){
  int i,j, index, count=0;
  int *id;
  double *buffer;

  // malloc buffer for sparse matrix's rows
  HANDLE_NULL( (buffer = (double *) malloc(N*sizeof(double))) );
  HANDLE_NULL( (id = (int *) malloc(N*sizeof(int))) );

  for (i=0; i<N; i++){
    // find neighbours of y[i] row
    gpu_calc_distances <<<blocks_per_grid, threads_per_block>>>(i,BANDWIDTH,d_buffer,d_y_data,d_x_data,d_sum);

    // get buffer from device
    HANDLE_ERROR( hipMemcpy(buffer, d_buffer, N*sizeof(double), hipMemcpyDeviceToHost) );
    
    // find neighbours (including diagonal elements)
    nNbr[i] = 0;
    for(j=0;j<N;j++)
      if(buffer[j]>0){
        id[nNbr[i]] = j;
        nNbr[i]++;
        count++;    // total elements of final sparse array
      }

    index = 0;
    HANDLE_NULL( (w[i] = (SparseData *) malloc(nNbr[i]*sizeof(SparseData))) );

    for (j=0; j<nNbr[i]; j++){
      // if (buffer[j] > 0){
        w[i][j].xid      = id[j];
        w[i][j].distance = buffer[id[j]];
        // index++;
      // }
    }
  }
  free(buffer);
  // here all the neighbours are known

  // move 2D host sparse to 1D device sparse
  HANDLE_ERROR( hipMalloc((void**) &d_sparse, count*sizeof(SparseData)) );

  int offset=0;
  for(i=0;i<N;i++){
    HANDLE_ERROR( hipMemcpy(&d_sparse[offset/sizeof(SparseData)], w[i], nNbr[i]*sizeof(SparseData), hipMemcpyHostToDevice) );
    offset += nNbr[i]*sizeof(SparseData);
  }

  // move nNbr to device
  HANDLE_ERROR( hipMemcpy(d_nNbr, nNbr, N*sizeof(int), hipMemcpyHostToDevice) );
}


__global__
void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData *w)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, i,j,sparse_offset=0;
  
  i = tid/D_SIZE;
  j = tid%D_SIZE;

  while(tid < N_SIZE*D_SIZE){
    //find the dynamic offset of rows that depend on the number of previous neighbours
    for (k=0;k<i;k++)
      sparse_offset += nNbr[k];
    
    y[tid] = 0;
    // multiply only the sparse element (all the other are 0's)
    for(k=0; k<nNbr[i]; k++)
      y[tid] += w[sparse_offset + k].distance * x[ (w[sparse_offset + k].xid * D_SIZE)/*row offset of x*/ + j ];

    tid += blockDim.x*gridDim.x;
  }
}

// TODO: reduction using shared memory
__global__ void gpu_normalize(int *nNbr, SparseData *w, double *y_new) 
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i,sparse_offset=0;
  double sum=0;
  
  i = tid/D_SIZE;

  while(tid < N_SIZE*D_SIZE) {
    //find the dynamic offset of rows that depend on the number of previous neighbours
    for (int k=0;k<i;k++)
      sparse_offset += nNbr[k];

    // TODO: sum can be shared if D>>2

    // find sum of current row
    for (int k=0; k<nNbr[i]; k++)
      sum += w[sparse_offset+k].distance; 

    y_new[tid] /= sum;
    tid += gridDim.x*blockDim.x;
  }
}

__global__ void gpu_calc_meanshift(double *m, double *y_new, double *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    m[tid] = y_new[tid] - y[tid];
    tid += gridDim.x+blockDim.x;
  }
}

__global__ void gpu_copy_2Darray(double *source, double *destination)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    destination[tid] = source[tid];
    tid += gridDim.x+blockDim.x;
  }
}


// TODO: non-shared implementation (use code from gpu_normalize)
__global__ void gpu_frob_norm_shared(double *m, double *result){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ float cache[threads_per_block];
  int cacheIndex = threadIdx.x;

  double norm = 0;
  while (tid < N_SIZE*D_SIZE) {
      norm += m[tid] * m[tid];
      tid += blockDim.x*gridDim.x;
  }
  
  // set the cache values
  cache[cacheIndex] = norm;
  
  // synchronize threads in this block
  __syncthreads();

  // for reductions, threads_per_block must be a power of 2
  int i = blockDim.x/2;
  while (i != 0) {
      if (cacheIndex < i)
        cache[cacheIndex] += cache[cacheIndex + i];

      __syncthreads();
      i /= 2;
  }

  // only 1rst thread of each block
  if (cacheIndex == 0)
      result[blockIdx.x] = cache[0];
}

// calculate last step of reduction on CPU because it's more efficient
double finish_reduction(){
  double *result, sum;

  // malloc result array
  HANDLE_NULL( (result = (double *) malloc(blocks_per_grid*sizeof(double))) );

  HANDLE_ERROR( hipMemcpy( result, 
                            d_sum,
                            blocks_per_grid*sizeof(float),
                            hipMemcpyDeviceToHost ) );
  sum = 0;
  for (int i=0; i<blocks_per_grid; i++)
      sum += result[i];
  
  free(result);
  return sum;
}

void print_2Darray(double **a, const int ROW, const int COL){
  int i,j;
  for (i=0;i<ROW;i++){
    for (j=0; j<COL; j++){
      printf("%lf \t",a[i][j]);
    }
  printf("\n");
  }
}
