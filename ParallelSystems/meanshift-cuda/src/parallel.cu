#include "hip/hip_runtime.h"
#include "../include/parallel.h"
#include "../include/global_vars.h"
#include "../include/cuda_helpers.h"

// constant device N and D
__device__ __constant__ int N_SIZE; 
__device__ __constant__ int D_SIZE;

// grid and block sizes
const int threads_per_block = 256;
const int blocks_per_grid = 32; 

typedef struct {
    int j;
    double distance;
} SparseData;

// host copies
double *x_data, *y_data, *buffer, **x, **y;
int *nNbr;
SparseData **w; 
SparseData **tmpW;

// device copies
double *d_x_data,*d_y_data,*d_y_new_data,*d_m_data,*d_sum, *d_buffer;
double **d_x,**d_y,**d_y_new,**d_m;
int *d_nNbr;
SparseData **d_w,*d_sparse; 


__global__ void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData *w);
__global__ void gpu_normalize(int *nNbr, SparseData *w, double *y_new);

extern "C"
void parallel(){
  printf("[INFO]: CUDA-GPU IMPLEMENTATION\n");
  printf("=============================\n");

  struct timeval startwtime, endwtime;
  double seq_time;

  init_parallel();
  
  gettimeofday (&startwtime, NULL);
  //------------------------------
  meanshift();
  //------------------------------
  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

  /*printf("\n\nIs test PASSed? %s\n\n", validate_parallel()?"YES":"NO");
  printf("===============================================\n\n");*/
  printf("\n\n[INFO]: Parallel meanshift wall clock time = %f\n", seq_time);

}

void init_parallel(){
  // blocks_per_grid = MIN(32, (N+threads_per_block-1) / threads_per_block);

  cpu_malloc();
  gpu_malloc();
  read_file();
  move_data_to_gpu();
}

//Contiguous memory allocation for 2D arrays
void cpu_malloc(){
  int i;
  
  if(VERBOSE) printf("[INFO]: Allocating cpu memory..\n");

  // malloc pointers to rows 
  HANDLE_NULL( (x = (double **) malloc(N * sizeof(double *))) );
  HANDLE_NULL( (y = (double **) malloc(N * sizeof(double *))) );
  HANDLE_NULL( (w = (SparseData **) malloc(N * sizeof(SparseData *))) );

  // malloc data of the arrays
  HANDLE_NULL( (x_data = (double *) malloc(N * D * sizeof(double))) );
  HANDLE_NULL( (y_data = (double *) malloc(N * D * sizeof(double))) );
  HANDLE_NULL( (nNbr = (int *) malloc(N * sizeof(int))) );

  // assign pointers of data to arrays
  for(i=0; i < N; i++){
    x[i]      = x_data      + i * D;
    y[i]      = y_data      + i * D;
  }
}


// Allocate memory for devive variables and copy x host data to device
void gpu_malloc (){
  int size; 

  if(VERBOSE) printf("[INFO]: Allocating device memory..\n");
  
  // malloc pointers of rows
  size = N * sizeof(double *);
  HANDLE_ERROR( hipMalloc((void**)&d_x,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m,    size) );


  // malloc data of the arrays
  size = N * D * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_x_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new_data,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m_data,    size) );

  // malloc d_sum
  size = blocks_per_grid * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_sum, size) );

  // malloc d_nNbr
  size = N * sizeof(int);
  HANDLE_ERROR( hipMalloc((void**)&d_nNbr, size) );

  // malloc d_buffer
  size = N * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_buffer, size) );

  // malloc d_w indexes of rows
  size = N * sizeof(SparseData *);
  HANDLE_ERROR( hipMalloc((void**)&d_w, size) );

    // get back indexes from device (need them in rangesearch2sparse)
  // HANDLE_ERROR( hipMemcpy(tmpW, d_w, sizeof(SparseData**), hipMemcpyDeviceToHost) );
}


void move_data_to_gpu(){
  if(VERBOSE) printf("[INFO]: Move data to device..\n");

  // move device constant variables N_SIZE and D_SIZE
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(N_SIZE), &N, sizeof(int)) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE), &D, sizeof(int)) );

  // move device global variable d_x and d_x_data
  HANDLE_ERROR( hipMemcpy(d_x,      x,      N*sizeof(double *), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_x_data, x_data, N*D*sizeof(double), hipMemcpyHostToDevice) );
}

void free_memory(){
  int i;
  if (VERBOSE) printf("[INFO]: Deallocating memory...\n");
  //free() data
  for (i=0; i<N; i++){
    free(x[i]);
    free(y[i]);
  }
  // free() pointers
  free(x);
  free(y);
}

void read_file(){
  int i,j;

  FILE * fp;
  fp = fopen (DATASET_PATH, "r");

  if (fp == NULL) { perror("[ERROR]: "); exit(1); }

  for (i=0; i<N; i++) 
    for (j=0; j<D; j++)
      if (EOF ==  fscanf(fp, "%lf", &x[i][j])) { perror("[ERROR]:"); exit(1); }

  fclose(fp);
}

void write_csv_file (char *message, double **a, const int ROW, const int COL){
  int i,j;

  FILE * fp;
  fp = fopen (OUTPUT_PATH, "w");

  if (fp == NULL){ perror("[ERROR]: "); exit(1); }

  fprintf(fp,"%s",message);

  for (i=0; i<ROW; i++) {
    for (j=0; j<COL; j++)
      if (EOF ==  fprintf(fp, "%lf, ", a[i][j])) {
        perror("[ERROR]:"); exit(1);
      }
    fprintf(fp,"\n");
  }

  fclose(fp);
}


void meanshift(){
  int iter=0;
  double norm = DBL_MAX;

  gpu_init_arr <<<blocks_per_grid, threads_per_block>>> (d_nNbr, d_x_data, d_y_data, d_m_data);

  
  while (norm > EPSILON){
    iter++;
    // find distances and calculate kernels
    rangesearch2sparse();

    // SparseData *test = (SparseData*) malloc(sum*sizeof(SparseData));

    // HANDLE_ERROR( hipMemcpy(test, d_sparse, sum*sizeof(SparseData), hipMemcpyDeviceToHost) );
    // int i,j;
    // for(i=0; i<5; i++)
    //   for(j=0; j<2; j++)
    //     printf("%f\n",y[i][j] );


    // compute new y vector
    gpu_matrix_mult <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_x_data,d_y_new_data,d_sparse);

    // normalize vector
    gpu_normalize <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_sparse,d_y_new_data);    

    // calculate meanshift
    gpu_calc_meanshift <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_y_new_data,d_y_data);

    // update y
    gpu_copy_2Darray <<<blocks_per_grid, threads_per_block>>>(d_y_new_data, d_y_data);

    // calculate Frobenius norm
    gpu_frob_norm_shared <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_sum);
    
    // //wait kernel calls to be executed
    // HANDLE_ERROR( hipDeviceSynchronize() );

    norm = sqrt ( finish_reduction() );

    if (VERBOSE) printf("[INFO]: Iteration %d - error %lf\n", iter, norm);   
    // exit(1); 
  }

  // copy results back to host
  if (VERBOSE){  
    HANDLE_ERROR( hipMemcpy(y_data, d_y_data, N*D*sizeof(double), hipMemcpyDeviceToHost) );
    write_csv_file("",y,N,D);
  }

  // gpu_free_memory();
  // cpu_free_memory();
}

// TODO: shared memory: the data within the block
__global__
void gpu_init_arr(int *nNbr, double *x, double *y, double *m)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;

  while (tid < N_SIZE*D_SIZE) {
    nNbr[tid%N_SIZE] = 0;
    y[tid] = x[tid];
    m[tid] = DBL_MAX;
    
    tid += blockDim.x * gridDim.x;
  }

}
// TODO: reduction with shared memory
__global__ void gpu_calc_distances
(int y_row, double h, double *buffer, double *y, double *x, double *n){

  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int i,j, x_arr_offset, y_arr_offset;
  
  double dist;

  while (tid < N_SIZE) {
    i = tid/D_SIZE;
    // diagonal elements
    if (y_row == tid){
      buffer[tid] = 1;
      tid += blockDim.x * gridDim.x;
      continue;
    }
    
    x_arr_offset = tid*D_SIZE;
    y_arr_offset = y_row*D_SIZE;  

    dist = 0; 
    for(j=0; j<D_SIZE; j++){
      dist += (y[y_arr_offset + j] - x[x_arr_offset + j])*(y[y_arr_offset + j] - x[x_arr_offset + j]);
    }

    // element inside radious
    if (dist < h*h){
      buffer[tid]= exp(-dist / (2.0*h*h));
    }
    // unnecessary elements
    else{
      buffer[tid]=0;
    }

    tid += blockDim.x * gridDim.x;
  }
}

void rangesearch2sparse(){
  int i,j, index, size;
  double *buffer;
  SparseData *tmp;

  // malloc buffer for sparse matrix's rows
  HANDLE_NULL( (buffer = (double *) malloc(N*sizeof(double))) );

  for (i=0; i<N; i++){
    // find neighbours of y[i] row
    gpu_calc_distances <<<blocks_per_grid, threads_per_block>>>(i,BANDWIDTH,d_buffer,d_y_data,d_x_data,d_sum);

    // number of neighbours
    // nNbr[i] = (int) finish_reduction(); 

    // get buffer from device
    HANDLE_ERROR( hipMemcpy(buffer, d_buffer, N*sizeof(double), hipMemcpyDeviceToHost) );
    
    // find neighbours
    nNbr[i] = 0;
    for(j=0;j<N;j++)
      if(buffer[j]>0) nNbr[i]+=1;   // include diagonal elements too

    // printf("\nnNbr[%d]=%d\n",i,nNbr[i]);
    


    index = 0;
    HANDLE_NULL( (w[i] = (SparseData *) malloc(nNbr[i]*sizeof(SparseData))) );

    for (j=0; j<N; j++){
      if (buffer[j] > 0){
        w[i][index].j        = j;
        w[i][index].distance = buffer[j];
        // printf("j = %d dist = %lf index = %d\n", tmp[index].j, tmp[index].distance, index);
        index++;
      }
    // printf("index = %d\n",index );
    }
  }


  int sum=0, offset;
  for(i=0;i<N;i++){
    sum += nNbr[i];
  }

  HANDLE_ERROR( hipMalloc((void**)&d_sparse, sum * sizeof(SparseData)) );

  // size_t free,total;
  // HANDLE_ERROR( hipMemGetInfo (&free, &total) );

  // printf("free = %zu total = %zu\n",free, total);

  sum=0;
  for(i=0;i<N;i++){
    offset = sum * sizeof(SparseData);
    // for (j=0;j<nNbr[i];j++){
    //   printf("%lf ", w[i][j].distance);
    // }
    // printf("%d %d\n",i, sum );
    HANDLE_ERROR( hipMemcpy(&d_sparse[sum], w[i], nNbr[i]*sizeof(SparseData), hipMemcpyHostToDevice) );
    sum += nNbr[i];
  }
  HANDLE_ERROR( hipMemcpy(d_nNbr, nNbr, N*sizeof(int), hipMemcpyHostToDevice) );

//==============================================================================
  SparseData *test = (SparseData*) malloc(sum*sizeof(SparseData));

  HANDLE_ERROR( hipMemcpy(test, d_sparse, sum*sizeof(SparseData), hipMemcpyDeviceToHost) );
  // int i,j;
  for(int i=0; i<nNbr[0]; i++){
      printf("%lf\n",w[0][i].distance );
      printf("%d\n",w[0][i].j );    
  }
//==============================================================================


  // SparseData *test = (SparseData*) malloc(sum*sizeof(SparseData));

  // HANDLE_ERROR( hipMemcpy(test, d_sparse, sum*sizeof(SparseData), hipMemcpyDeviceToHost) );
  // for (i=0;i<sum;i++){
  //   printf("%f \n", test[i].distance);
  // }

  // printf("das\n");
  // exit(1);
   /* // printf("%p\n", tmp);
    // printf("%p\n", d_w[i]);
    // printf("%d\n", nNbr[i]*sizeof(SparseData));
    printf("Nbr[%d]=%d\n",i,nNbr[i]);

    // malloc w[i] device memory
    HANDLE_ERROR( hipMalloc((void**)&w[i], nNbr[i] * sizeof(SparseData)) );
    // move value of d_w[i] to device
    HANDLE_ERROR( hipMemcpy(&tmpW[i], &w[i], sizeof(SparseData*), hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMemcpy(&  tmpW[i], tmp, nNbr[i]*sizeof(SparseData), hipMemcpyHostToDevice) );

    free (tmp);*/
  // }

  // free(buffer);
}


__global__
void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData *w)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, offset, i,j,sparse_offset=0;
  
  i = tid/D_SIZE;
  j = tid%D_SIZE;

  while(tid < N_SIZE*D_SIZE){

    for (k=0;k<i;k++)
      sparse_offset += nNbr[k];
    
    
    offset = i*D_SIZE;
    y[tid] = 0;
    for(k=0; k<nNbr[i]; k++)
      y[tid] += w[sparse_offset + k].distance * x[ w[sparse_offset + k].j * D_SIZE + j ];

    tid += blockDim.x*gridDim.x;

    if(tid == 0)
      for (int l=0; l<10; l++)
        printf("%lf\n",y[l]);
      
  }
}

// TODO: reduction using shared memory
__global__ void gpu_normalize(int *nNbr, SparseData *w, double *y_new) 
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,sparse_offset=0;
  double sum=0;
  
  i = tid/D_SIZE;
  j = tid%D_SIZE;

  while(tid < N_SIZE*D_SIZE) {
    for (int j=0;j<i;j++){
      sparse_offset += nNbr[j];
    }

    for (int k=0; k<nNbr[i]; k++)
      sum += w[sparse_offset+k].distance; // not contiguous memory, [][] necessary

    y_new[tid] /= sum;
    tid += gridDim.x*blockDim.x;
  }
}

__global__ void gpu_calc_meanshift(double *m, double *y_new, double *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    m[tid] = y_new[tid] - y[tid];
    tid += gridDim.x+blockDim.x;
  }
}

__global__ void gpu_copy_2Darray(double *source, double *destination)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    destination[tid] = source[tid];
    tid += gridDim.x+blockDim.x;
  }
}

/*// TODO: non-shared implementation (use code from gpu_normalize)
__global__ void gpu_frob_norm(double *m, double *result){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  double norm = 0;
  while (tid < N_SIZE*D_SIZE) {
      norm += m[tid] * m[tid];
      tid += blockDim.x*gridDim.x;
  }
  
  // set the cache values
  cache[cacheIndex] = norm;
  
  // synchronize threads in this block
  __syncthreads();

  // for reductions, threads_per_block must be a power of 2
  int i = blockDim.x/2;
  while (i != 0) {
      if (cacheIndex < i)
          cache[cacheIndex] += cache[cacheIndex + i];
      __syncthreads();
      i /= 2;
  }

  if (cacheIndex == 0)
      result[blockIdx.x] = cache[0];
}*/

// TODO: non-shared implementation (use code from gpu_normalize)
__global__ void gpu_frob_norm_shared(double *m, double *result){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ float cache[threads_per_block];
  int cacheIndex = threadIdx.x;

  double norm = 0;
  while (tid < N_SIZE*D_SIZE) {
      norm += m[tid] * m[tid];
      tid += blockDim.x*gridDim.x;
  }
  
  // set the cache values
  cache[cacheIndex] = norm;
  
  // synchronize threads in this block
  __syncthreads();

  // for reductions, threads_per_block must be a power of 2
  int i = blockDim.x/2;
  while (i != 0) {
      if (cacheIndex < i)
        cache[cacheIndex] += cache[cacheIndex + i];

      __syncthreads();
      i /= 2;
  }

  if (cacheIndex == 0)
      result[blockIdx.x] = cache[0];
}

// calculate last step of reduction on CPU because it's more efficient
double finish_reduction(){
  double *result, sum;

  // malloc result array
  HANDLE_NULL( (result = (double *) malloc(blocks_per_grid*sizeof(double))) );

  HANDLE_ERROR( hipMemcpy( result, 
                            d_sum,
                            blocks_per_grid*sizeof(float),
                            hipMemcpyDeviceToHost ) );
  sum = 0;
  for (int i=0; i<blocks_per_grid; i++){
      sum += result[i];
  }
  free(result);
  return sum;
}

void print_2Darray(double **a, const int ROW, const int COL){
  int i,j;
  for (i=0;i<ROW;i++){
    for (j=0; j<COL; j++){
      printf("%lf \t",a[i][j]);
    }
  printf("\n");
  }
}
