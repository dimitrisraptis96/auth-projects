#include "hip/hip_runtime.h"
#include "../include/parallel.h"
#include "../include/global_vars.h"
#include "../include/kernels.cuh"

// constant device N and D
__device__ __constant__ int N_SIZE; 
__device__ __constant__ int D_SIZE;

// grid and block sizes
const int threads_per_block = 256;
const int blocks_per_grid = 32; 

typedef struct {
    int xid;
    double distance;
} SparseData;

// host copies 
double *arr_data, **arr;

// device copies
double *d_x_data,*d_y_data,*d_y_new_data,*d_m_data,*d_reduction, *d_sum, *d_Pdist;
int *d_nNbr;
SparseData *d_sparse; 

// declare them here because they include SparseData struct
__global__ void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData *w);
__global__ void gpu_normalize(int *nNbr, SparseData *w, double *y_new, double *sum);

extern "C"
void parallel(){
  printf("===============================\n");
  printf("[INFO]: CUDA-GPU IMPLEMENTATION\n");
  printf("===============================\n");
  printf("[INFO]: bandwidth=%lf\n",BANDWIDTH);
  printf("[INFO]: epsilon=%lf\n\n",EPSILON);

  struct timeval startwtime, endwtime;
  double seq_time;

  // choose exhaustive or sparse version
  int version = choose_version();
  init_parallel(version);
  
  gettimeofday (&startwtime, NULL);
  //------------------------------
  cuda_meanshift(version);
  //------------------------------
  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

  printf("\n\n[FINAL]: parallel meanshift wall clock time = %f\n\n", seq_time);
}

void init_parallel(int version){
  if (VERBOSE) 
    printf ("[INFO]: VERSION: %s & %s\n\n", 
          (version==VERSION_EXHAUSTIVE) ? "exhaustive": "sparse",
          (USE_SHARED)                  ? "shared"    : "non-shared");
  
  cpu_malloc();
  gpu_malloc(version);
  read_file();
  move_data_to_gpu();
}

// choose version according to N value and global memory size
int choose_version(){
  // get device props
  hipDeviceProp_t  prop;
  HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
  
  double bytes = prop.totalGlobalMem / sizeof(double);

  return (N*N > bytes/2) ? VERSION_SPARSE : VERSION_EXHAUSTIVE; 
}

// ====================================================================
// ====================================================================
//                      MEMORY ALLOCATION
// ====================================================================
// ====================================================================

//Contiguous memory allocation for 2D array
void cpu_malloc(){
  if(VERBOSE) printf("[INFO]: allocate cpu memory..\n");

  // malloc pointers to rows 
  HANDLE_NULL( (arr = (double **)     malloc(N * sizeof(double *))) );

  // malloc data of the array
  HANDLE_NULL( (arr_data = (double *) malloc(N * D * sizeof(double))) );

  // assign pointers of data to array
  int i;
  for(i=0; i < N; i++){
    arr[i]      = arr_data      + i * D;
  }
}


// Allocate memory for devive arrays
void gpu_malloc (int version){
  int size; 

  if(VERBOSE) printf("[INFO]: allocate device memory..\n");

  if(version == VERSION_EXHAUSTIVE){
    // malloc d_Pdist
    size = N * N * sizeof(double);
    HANDLE_ERROR( hipMalloc((void**)&d_Pdist, size) );
  }

  // malloc data of the arrays
  size = N * D * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_x_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new_data,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m_data,    size) );

  // malloc d_reduction
  size = blocks_per_grid * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_reduction, size) );

  // malloc d_sum
  size = N * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_sum, size) );

  // malloc d_nNbr
  size = N * sizeof(int);
  HANDLE_ERROR( hipMalloc((void**)&d_nNbr, size) );
}


void move_data_to_gpu(){
  if(VERBOSE) printf("[INFO]: move data to device..\n\n");

  // move to device constant variables N_SIZE and D_SIZE
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(N_SIZE), &N, sizeof(int)) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE), &D, sizeof(int)) );

  // move to device x array
  HANDLE_ERROR( hipMemcpy(d_x_data, arr_data, N*D*sizeof(double), hipMemcpyHostToDevice) );
}

void cpu_free_memory(){
  if (VERBOSE) printf("[INFO]: deallocate cpu memory...\n");
  
  // free global memory
  free(arr);
  free(arr_data);
}

void gpu_free_memory(int version){
  if (VERBOSE) printf("\n[INFO]: deallocate gpu memory...\n");

  // free gpu memory
  HANDLE_ERROR( hipFree(d_x_data) );
  HANDLE_ERROR( hipFree(d_y_data) );
  HANDLE_ERROR( hipFree(d_m_data) );
  HANDLE_ERROR( hipFree(d_y_new_data) );
  HANDLE_ERROR( hipFree(d_reduction) );
  HANDLE_ERROR( hipFree(d_nNbr) );
  switch (version){
    case VERSION_EXHAUSTIVE:
      HANDLE_ERROR( hipFree(d_Pdist) );
      break;

    case VERSION_SPARSE:
      HANDLE_ERROR( hipFree(d_sparse) );
      break;
  }
}

// ====================================================================
// ====================================================================
//                      I/O OPERATIONS
// ====================================================================
// ====================================================================

void read_file(){
  int i,j;

  FILE * fp;
  HANDLE_NULL( (fp = fopen (DATASET_PATH, "r")) );

  for (i=0; i<N; i++) 
    for (j=0; j<D; j++)
      HANDLE_EOF( (fscanf(fp, "%lf", &arr[i][j])) );

  HANDLE_EOF( (fclose(fp)) );
}

void write_csv_file (char *message, double **a, const int ROW, const int COL){
  int i,j;

  FILE * fp;
  HANDLE_NULL( (fp = fopen (OUTPUT_PATH_PARALLEL, "w")) );

  if (message != NULL)  HANDLE_EOF( (fprintf(fp,"%s",message)) );

  for (i=0; i<ROW; i++) {
    for (j=0; j<COL; j++){
      if (j == COL-1){
        HANDLE_EOF( fprintf(fp, "%lf", a[i][j]) );
      } 
      else {
        HANDLE_EOF( fprintf(fp, "%lf, ", a[i][j]) ); 
      }
    }
    HANDLE_EOF( fprintf(fp,"\n") );
  }

  HANDLE_EOF( (fclose(fp)) );
}

// ====================================================================
// ====================================================================
//                      MEANSHIFT IMPLEMENTATION
// ====================================================================
// ====================================================================

void cuda_meanshift(int version){

  int iter=0;
  double norm = DBL_MAX;

  gpu_init_arr <<<blocks_per_grid, threads_per_block>>> (d_x_data, d_y_data, d_m_data);

  
  while (norm > EPSILON){
    iter++;

    switch(version){

      case VERSION_EXHAUSTIVE:

        // find distances and calculate kernels
        gpu_pdist<<<blocks_per_grid, threads_per_block>>>(BANDWIDTH,d_Pdist, d_y_data, d_x_data );

        // compute new y vector
        gpu_matrix_mult_exh <<<blocks_per_grid, threads_per_block>>>(d_x_data,d_y_new_data,d_Pdist);

        // normalize vector
        gpu_normalize_exh <<<blocks_per_grid, threads_per_block>>>(d_y_new_data,d_Pdist);    
        
        break;

      case VERSION_SPARSE:
        
        // find distances and calculate kernels
        rangesearch2sparse();

        // compute new y vector
        gpu_matrix_mult <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_x_data,d_y_new_data,d_sparse);

        // normalize vector
        gpu_normalize <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_sparse,d_y_new_data,d_sum);    
        
        break;
    }

    // calculate meanshift
    gpu_calc_meanshift <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_y_new_data,d_y_data);
    
    // update y
    gpu_copy_2Darray <<<blocks_per_grid, threads_per_block>>>(d_y_new_data, d_y_data);
    
    // calculate Frobenius norm
    (USE_SHARED) ? 
          // shared norm
          gpu_frob_norm_shared <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_reduction):
          // non-shared norm
          gpu_frob_norm        <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_sum);

    norm = sqrt ( finish_reduction() );

    if (VERBOSE) printf("[INFO]: iteration %d - error %lf\n", iter, norm);   
  }

  // copy results back to host
  if (VERBOSE){  
    HANDLE_ERROR( hipMemcpy(arr_data, d_y_new_data, N*D*sizeof(double), hipMemcpyDeviceToHost) );
    write_csv_file(NULL,arr,N,D);
  }

  gpu_free_memory(version);
  cpu_free_memory();
}

// initialize the device arrays
__global__
void gpu_init_arr(double *x, double *y, double *m)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;

  while (tid < N_SIZE*D_SIZE) {
    y[tid] = x[tid];
    m[tid] = DBL_MAX;
    
    tid += blockDim.x * gridDim.x;
  }
}


// ====================================================================
// ====================================================================
//                      EXHAUSTIVE VERSION
// ====================================================================
// ====================================================================

// calculate the distances between x and y points
__global__ 
void gpu_pdist(double h, double *out, double *y, double *x)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  double tmp, dist;
  
  while (tid < N_SIZE) {

    for(int r=0; r<N_SIZE; r++){ //outer loop
      dist = 0;

     // calculate distances of y[r] point and all x[tid] points
     for(int i=0; i<D_SIZE; i++){
        tmp = y[r*D_SIZE +i] - x[tid*D_SIZE + i];
        dist += tmp*tmp;
      }

      // y is rows and x is columns
      if (dist == 0) {
        // diagonal elements
        out[r*N_SIZE+tid] = 1;
      }
      else if(dist < h*h){
        // distances inside radius
        out[r*N_SIZE+tid] = exp(-dist / (2.0*h*h));
      }
      else {
        // unnecessary distances
        out[r*N_SIZE+tid] = 0;
      }
    }

    tid += gridDim.x*blockDim.x;
  }
}

__global__
void gpu_matrix_mult_exh(double *x, double *y, double *dist)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, i,j;
  
  while(tid < N_SIZE*D_SIZE){
    i = tid/D_SIZE; // i between [0,N_SIZE-1]
    j = tid%D_SIZE; // j between [0,D_SIZE-1]
    
    y[tid] = 0;

    for(k=0; k<N_SIZE; k++)
      y[tid] += dist[i*N_SIZE + k] * x[k*D_SIZE + j];

    tid += blockDim.x*gridDim.x;
  }
}


__global__ void gpu_normalize_exh(double *y_new, double *dist) 
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i;
  double sum;

  while(tid < N_SIZE*D_SIZE) {
    i = tid/D_SIZE;

    // calculate sum of the array
    sum = 0;
    for (int k=0; k<N_SIZE; k++)
      sum += dist[i*N_SIZE + k]; 
    // normalize here
    y_new[tid] /= sum;

    tid += gridDim.x*blockDim.x;
  }
}

// ====================================================================
// ====================================================================
//                      SPARSE VERSION
// ====================================================================
// ====================================================================

void rangesearch2sparse(){
  int i,j, count=0;
  int *id, *nNbr;
  double *buffer, *sum, *d_buffer;
  SparseData **w;

  // malloc host arrays
  HANDLE_NULL( (buffer  = (double *) malloc(N * sizeof(double))) ); // distances of y[i] row
  HANDLE_NULL( (sum     = (double *) malloc(N * sizeof(double))) ); // sum of i row
  HANDLE_NULL( (id      = (int *)    malloc(N * sizeof(int))) );    // id's of distances inside bandwidth
  HANDLE_NULL( (nNbr    = (int *)    malloc(N * sizeof(int))) );    // number of neighbours of y[i] row
  HANDLE_NULL( (w  = (SparseData **) malloc(N * sizeof(SparseData *))) );

  // malloc device array
  HANDLE_ERROR( hipMalloc((void**)&d_buffer, N * sizeof(double)) );

  for (i=0; i<N; i++){
    // find neighbours of y[i] row
    gpu_calc_distances <<<blocks_per_grid, threads_per_block>>>(i,BANDWIDTH,d_buffer,d_y_data,d_x_data);

    // get buffer from device
    HANDLE_ERROR( hipMemcpy(buffer, d_buffer, N*sizeof(double), hipMemcpyDeviceToHost) );
    
    // find neighbours (including diagonal elements)
    sum[i]=0;
    nNbr[i] = 0; 
    for(j=0;j<N;j++)
      if(buffer[j]>0){
        sum[i] += buffer[j];  //total dist sum of y[i] row
        id[nNbr[i]] = j;
        nNbr[i]++;
        count++;    // total elements of final sparse array
      }
    // here all the neighbours are known!

    HANDLE_NULL( (w[i] = (SparseData *) malloc(nNbr[i]*sizeof(SparseData))) );

    // nNbr[i] << N
    for (j=0; j<nNbr[i]; j++){
        w[i][j].xid      = id[j];
        w[i][j].distance = buffer[id[j]];
    }
  }
  HANDLE_ERROR( hipFree(d_buffer) );
  // here sparse is ready!!!
  
  // making contiguous sparse eliminates most of the per-transfer overhead
  SparseData *tmp_sparse;
  HANDLE_NULL( (tmp_sparse = (SparseData *) malloc(count*sizeof(SparseData))) );
  int index=0;
  for(i=0;i<N;i++){
    for(j=0;j<nNbr[i];j++){
      tmp_sparse[index] = w[i][j];
      index++;
    }
  }

  // move 2D host sparse to 1D device sparse
  HANDLE_ERROR( hipFree(d_sparse) ); // need these for iterations after the 1rst one
  HANDLE_ERROR( hipFree(d_sum) );
  HANDLE_ERROR( hipFree(d_nNbr) );
  HANDLE_ERROR( hipMalloc((void**) &d_sparse, count*sizeof(SparseData)) );
  HANDLE_ERROR( hipMemcpy(d_sparse, tmp_sparse, count*sizeof(SparseData), hipMemcpyHostToDevice) );

  // move nNbr to device
  HANDLE_ERROR( hipMemcpy(d_nNbr, nNbr, N*sizeof(int),    hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_sum,  sum,  N*sizeof(double), hipMemcpyHostToDevice) );
  
  // free host memory
  for(i=0;i<N;i++)
    free(w[i]);
  free(tmp_sparse);
  free(nNbr); free(sum); free(w); free(buffer); free(id);
}
            
__global__ void gpu_calc_distances
(int y_row, double h, double *buffer, double *y, double *x){

  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, x_arr_offset, y_arr_offset;
  
  double dist;

  while (tid < N_SIZE) {
    // diagonal elements
    if (y_row == tid){
      buffer[tid] = 1;
      tid += blockDim.x * gridDim.x;
      continue;
    }
    
    x_arr_offset = tid*D_SIZE;
    y_arr_offset = y_row*D_SIZE;  

    // calculate distances
    dist = 0; 
    for(k=0; k<D_SIZE; k++){
      dist += (y[y_arr_offset + k] - x[x_arr_offset + k])*(y[y_arr_offset + k] - x[x_arr_offset + k]);
    }

    // element inside radious
    if (dist < h*h){
      buffer[tid]= exp(-dist / (2.0*h*h));
    }
    // unnecessary elements
    else{
      buffer[tid]=0;
    }

    tid += blockDim.x * gridDim.x;
  }
}


__global__
void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData *w)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, i,j,sparse_offset;
  

  while(tid < N_SIZE*D_SIZE){
    // i and j indexes of flattened 2D array x
    i = tid/D_SIZE;
    j = tid%D_SIZE;
    
    //find the dynamic offset of rows that depend on the number of previous neighbours
    sparse_offset=0;
    for (k=0;k<i;k++)
      sparse_offset += nNbr[k];
    
    y[tid] = 0;
    // multiply only the sparse element (all the other are 0's)
    for(k=0; k<nNbr[i]; k++)
      y[tid] += w[sparse_offset + k].distance * x[ (w[sparse_offset + k].xid * D_SIZE)/*row offset of x*/ + j ];

    tid += blockDim.x*gridDim.x;
  }
}


__global__ void gpu_normalize(int *nNbr, SparseData *w, double *y_new, double *sum) 
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i;

  while(tid < N_SIZE*D_SIZE) {
    i = tid/D_SIZE;

    y_new[tid] /= sum[i];
    tid += gridDim.x*blockDim.x;
  }
}

// ====================================================================
// ====================================================================
//                      SAME FOR BOTH VERSIONS
// ====================================================================
// ====================================================================

__global__ void gpu_calc_meanshift(double *m, double *y_new, double *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    m[tid] = y_new[tid] - y[tid];
    tid += gridDim.x*blockDim.x;
  }
}

__global__ void gpu_copy_2Darray(double *src, double *dst)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    dst[tid] = src[tid];
    tid += gridDim.x*blockDim.x;
  }
}

__global__ void gpu_frob_norm(double *m, double *final){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N_SIZE){
    double sum = 0;
    for (int i=0;i<D_SIZE;i++){
      sum += m[tid*D_SIZE+i]*m[tid*D_SIZE+i];
    }
    final[tid] = sum; // here final's size is N

    tid += blockDim.x*gridDim.x;
  }
}

__global__ void gpu_frob_norm_shared(double *m, double *final){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ double cache[threads_per_block];
  int cacheIndex = threadIdx.x;

  double norm = 0;
  while (tid < N_SIZE*D_SIZE) {
      norm += m[tid] * m[tid];
      tid += blockDim.x*gridDim.x;
  }
  
  // set the cache values
  cache[cacheIndex] = norm;
  
  // synchronize threads in this block
  __syncthreads();

  // for reductions, threads_per_block must be a power of 2
  int i = blockDim.x/2;
  while (i != 0) {
      if (cacheIndex < i)
        cache[cacheIndex] += cache[cacheIndex + i];

      __syncthreads();
      i /= 2;
  }

  // only 1rst thread of each block
  if (cacheIndex == 0)
    final[blockIdx.x] = cache[0]; //here final's size is blocks_per_grid
}


// calculate last step of reduction on CPU because it's more efficient
double finish_reduction(){
  double  sum, *result, *device;
  int size;

  // decide the size and the device array according to shared/non-shared option
  if (USE_SHARED) {
    size   = blocks_per_grid;
    device = d_reduction;
  } 
  else {
    size   = N;
    device = d_sum;
  }

  HANDLE_NULL( (result = (double *) malloc(size * sizeof(double))) );

  HANDLE_ERROR( hipMemcpy( result,
                            device,
                            size*sizeof(double),
                            hipMemcpyDeviceToHost ) );
  sum = 0;
  for (int i=0; i<size; i++){
      sum += result[i];
  }
  free(result);
  return sum;
}

void print_2Darray(double **a, const int ROW, const int COL){
  int i,j;
  for (i=0;i<ROW;i++){
    for (j=0; j<COL; j++){
      printf("%lf \t",a[i][j]);
    }
  printf("\n");
  }
}