#include "hip/hip_runtime.h"
#include "../include/global_vars.h"
#include "../include/parallel.h"
#include "../include/helpers.h"

// TODO: prepare_gpu()
// TODO: malloc continuous memory !!!

__global__ 
void init_arr(int *d_nNbr,double *d_x_data, double *d_y_data, double *d_m_data);
void cuda_error_handler(hipError_t err);

__device__ int GRID_SIZE
__device__ int BLOCK_SIZE

typedef struct {
    int j;
    double distance;
} SparseData;

// host copies
  double **x, **y;
  double *x_data, *y_data;

// device copies
  double **d_x,**d_y,**d_y_new,**d_m;
  double *d_x_data,*d_y_data,*d_y_new_data,*d_m_data;
  int *d_nNbr;
  SparseData **d_w;  
  int *limit; 


extern "C"
void parallel(){
  printf("[INFO]: CUDA-GPU IMPLEMENTATION\n");
  printf("=============================\n");

  struct timeval startwtime, endwtime;
  double seq_time;

  init_parallel();
  
  gettimeofday (&startwtime, NULL);
  //------------------------------
  meanshift();
  //------------------------------
  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

  /*printf("\n\nIs test PASSed? %s\n\n", validate_parallel()?"YES":"NO");
  printf("===============================================\n\n");*/
  printf("\n\n[INFO]: Parallel meanshift wall clock time = %f\n", seq_time);

}

// Check if cuda API calls returned successfully
void cuda_error_handler(hipError_t err){
  if (err != hipSuccess) {
    printf("%s\n", hipGetErrorString(err));
    exit(1);
  }
}

void init_parallel(){
  // define device global variables
  cuda_error_handler( cudaMemCpyToSymbol (GRID_SIZE,  &N, sizeof(int), 0, cudaCpyHostToDevice) );
  cuda_error_handler( cudaMemCpyToSymbol (BLOCK_SIZE, &D, sizeof(int), 0, cudaCpyHostToDevice) );

  cpu_malloc();
  gpu_malloc();
  read_file();
  move_data_to_gpu();
}

//Contiguous memory allocation for 2D arrays
void cpu_malloc(){
  int i;
  
  if(VERBOSE) printf("[INFO]: Allocating cpu memory..\n");

  // malloc pointers to rows 
  x     = (double **) malloc(N * sizeof(double *));
  y     = (double **) malloc(N * sizeof(double *));

  if (x == NULL || y == NULL) {perror("[ERROR]:"); exit(1);} 

  // malloc data of the arrays
  x_data      = (double *) malloc(N * D * sizeof(double));
  y_data      = (double *) malloc(N * D * sizeof(double));

  if(x_data == NULL || y_data == NULL) {perror("[ERROR]:"); exit(1);}

  // assign pointers of data to arrays
  for(i=0; i < N; i++){
    x[i]      = x_data      + i * D;
    y[i]      = y_data      + i * D;
  }
}


// Allocate memory for devive variables and copy x host data to device
void gpu_malloc (){
  int size; 

  if(VERBOSE) printf("[INFO]: Allocating device memory..\n");
  
  // malloc pointers of rows
  size = N * sizeof(double *);
  cuda_error_handler( hipMalloc((void**)&d_x,    size) );
  cuda_error_handler( hipMalloc((void**)&d_y,    size) );
  cuda_error_handler( hipMalloc((void**)&d_y_new,size) );
  cuda_error_handler( hipMalloc((void**)&d_m,    size) );

  // malloc data of the arrays
  size = N * D * sizeof(double);
  cuda_error_handler( hipMalloc((void**)&d_x_data,    size) );
  cuda_error_handler( hipMalloc((void**)&d_y_data,    size) );
  cuda_error_handler( hipMalloc((void**)&d_y_new_data,size) );
  cuda_error_handler( hipMalloc((void**)&d_m_data,    size) );

  //malloc nNbr
  size = N * sizeof(int);
  cuda_error_handler( hipMalloc((void**)&d_nNbr, size) );

  //malloc w indexes of rows
  size = N * sizeof(SparseData *);
  cuda_error_handler( hipMalloc((void**)&d_w, size) );

  // int value
  size = sizeof(int);
  cuda_error_handler( hipMalloc((void**)&limit, size) );
}

void move_data_to_gpu(){
  int size;

  if(VERBOSE) printf("[INFO]: Move data to device..\n");

  size = N * sizeof(double *);
  cuda_error_handler( hipMemcpy(d_x, x, size, hipMemcpyHostToDevice) );
  size = N * D * sizeof(double);
  cuda_error_handler( hipMemcpy(d_x_data, x_data, size, hipMemcpyHostToDevice) );
}

void free_memory(){
  int i;
  if (VERBOSE) printf("[INFO]: Deallocating memory...\n");
  //free() data
  for (i=0; i<N; i++){
    free(x[i]);
    free(y[i]);
  }
  // free() pointers
  free(x);
  free(y);
}

void read_file(){
  int i,j;

  FILE * fp;
  fp = fopen (DATASET_PATH, "r");

  if (fp == NULL) { perror("[ERROR]: "); exit(1); }

  for (i=0; i<N; i++) 
    for (j=0; j<D; j++)
      if (EOF ==  fscanf(fp, "%lf", &x[i][j])) { perror("[ERROR]:"); exit(1); }

  fclose(fp);
}

void write_csv_file (char *message, double **a, const int ROW, const int COL){
  int i,j;

  FILE * fp;
  fp = fopen (OUTPUT_PATH, "w");

  if (fp == NULL){ perror("[ERROR]: "); exit(1); }

  fprintf(fp,"%s",message);

  for (i=0; i<ROW; i++) {
    for (j=0; j<COL; j++)
      if (EOF ==  fprintf(fp, "%lf, ", a[i][j])) {
        perror("[ERROR]:"); exit(1);
      }
    fprintf(fp,"\n");
  }

  fclose(fp);
}

__global__ 
void init_arr(int *limit, 
              int *d_nNbr, 
              double *d_x_data, 
              double *d_y_data, 
              double *d_m_data)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  
  // TODO: shared memory: the data within the block

  if (id < GRID_SIZE*BLOCK_SIZE) {
    if(id%2 == 0) d_nNbr[id/2] = 0; // can be optized
    d_y_data[id] = d_x_data[id];
    d_m_data[id] = DBL_MAX;
  }
}

/*void set_threads(int size){

  if ((size) < 32) {
    threads_num = size;
    blocks_num  = 1;
  }
  else {
    threads_num = size;
    blocks_num  = ceil(float(size / threads_num);
  }
}
*/


void meanshift(){
  int iter=0;
  int nblocks=N, nthreads=D;
  double norm = DBL_MAX;

  init_arr <<<N,D>>> (limit, d_nNbr, d_x_data, d_y_data, d_m_data); ///not ready!!!!!!!!

  cuda_error_handler( hipMemcpy(y_data, d_y_data, N * D * sizeof(double), hipMemcpyDeviceToHost) );

  write_csv_file("",y,N,D);

  /*while (norm > EPSILON){
    iter++;

    // find distances between each row of y and the rows of x that are BANDWIDTH or less distant.
    // And calculate kernels for these distances.
    rangesearch2sparse();

    // compute new y vector
    matrix_mult();

    // normalize vector
    normalize();    

    // calculate meanshift
    calc_meanshift();

    // update y
    copy_2Darray(y_new, y,N,D);

    // calculate Frobenius norm
    norm = frob_norm();

    if (VERBOSE){
      printf("[INFO]: Iteration %d - error %lf\n", iter, norm);
    }
  } 
  if (VERBOSE)  write_csv_file("",y_new,N,D);*/
  
  // free_memory();
}


// void rangesearch2sparse(){
//   int i,j, index;
//   double dist;

//   // malloc buffer for sparse matrix's rows
//   double *buffer = (double *) malloc(N*sizeof(double));
//   if(buffer == NULL) { perror("[ERROR]:");exit(1); }

//   for (i=0; i<N; i++){
//     for (j=0; j<N; j++){
//       // make sure diagonal elements are 1
//       if (i==j) {
//         buffer[j] = 1; nNbr[i]++; 
//         continue;
//       }

//       // find distances inside radius
//       dist = euclidean_distance(i,j);
//       if (dist < BANDWIDTH*BANDWIDTH){  // radius^2 because I don't use sqrt() at dist
//         buffer[j]= gaussian_kernel(dist);
//         nNbr[i]++;
//       }
//       // unnecessary points
//       else{
//         buffer[j]=0;
//       }
//     }

//     // malloc sparse matrix (w) rows
//     w[i]  = (SparseData *) malloc(nNbr[i] * sizeof(SparseData));
//     if(w[i]==NULL) {perror("[ERROR]: "); exit(1);}

//     index = 0;
//     for (j=0; j<N; j++){
//       if (buffer[j] > 0){
//         w[i][index].j        = j;
//         w[i][index].distance = buffer[j]; 
//         index++;
//       }
//     }
//   }
// }

// /*__global__ void matrix_mult(int *d_nNbr, double **d_y_new, SparseData **d_w)
// {
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   int idy = blockIdx.y * blockDim.y + threadIdx.y;
//   int k;

//   if((idx < N) && (idy < D)) {
//     y_new[idx][idy] = 0;
//     for(k=0; k<d_nNbr[i]; k++)
//         d_y_new[idx][idy] += d+w[idx][k].distance * x[ w[idx][k].j ][idy];
//   }
// }*/

// void matrix_mult() {
//   int i,j,k;
//   for(i=0; i<N; i++){
//     for(j=0; j<D; j++){
//       y_new[i][j] = 0;
//       for(k=0; k<nNbr[i]; k++)
//           y_new[i][j] += w[i][k].distance * x[ w[i][k].j ][j];
//     }
//   }
// }

// /*__global__ void normalize(int *d_nNbr, double **d_y_new, SparseData **d_w)
// {
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   int idy = blockIdx.y * blockDim.y + threadIdx.y;
//   double sum=0;  //shared within block for optimization

//   if((idx < N) && (idy < D)) {
//     if (threadIdx.x == 0) sum = sum_of_row(i);
//     d_y_new[idx][idy] /= sum;
//   }
// }*/


// __device__ double sum_of_row(const int row_index){
//   // TODO call this from device
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   double sum=0;
  
//   if(idx < nNbr[row_index])
//     sum += w[row_index][j].distance;
//   __synchronized //wait all threads to sum the
//   return sum; // make sure it returns the correct sum
// }

// void normalize(){
//   int i,j;
//   double s=0;

//   for (i=0;i<N;i++){
//     s = sum_of_row(i);
//     for (j=0; j<D; j++)
//       y_new[i][j] /= s;       
//   }
// }

// double sum_of_row(const int row_index){
//   int j;
//   double sum=0;
  
//   for (j=0; j<nNbr[row_index]; j++)
//     sum += w[row_index][j].distance;
//   return sum;
// }

// /*__global__ double frob_norm(int *d_nNbr, double **d_y_new, SparseData **d_w)
// {
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   int idy = blockIdx.y * blockDim.y + threadIdx.y;
//   double sum=0;  //shared within block for optimization

//   if((idx < N) && (idy < D)) {
//     if (threadIdx.x == 0) sum = sum_of_row(i);
//     d_y_new[idx][idy] /= sum;
//   }
// }*/


// double frob_norm(){
//   int i,j;
//   double norm=0;
//   for (i=0; i<N; i++)
//     for (j=0; j<D; j++)
//       norm += m[i][j] * m[i][j];
//   return sqrt(norm);
// }

// void calc_meanshift(){
//   int i,j;
//   for (i=0;i<N;i++)
//     for (j=0; j<D; j++)
//       m[i][j] = y_new[i][j] - y[i][j];       
// }

// void copy_2Darray(double **source, double **destination, const int ROW, const int COL){
//   int i,j;
//   for (i=0;i<ROW;i++)
//     for (j=0; j<COL; j++)
//       destination[i][j] = source[i][j];
// }

void print_2Darray(double **a, const int ROW, const int COL){
  int i,j;
  for (i=0;i<ROW;i++){
    for (j=0; j<COL; j++){
      printf("%lf \t",a[i][j]);
    }
  printf("\n");
  }
}

// double gaussian_kernel(const double dist){
//     return exp(- dist / (2.0*BANDWIDTH*BANDWIDTH));
// }

// double euclidean_distance(const int first, const int second){
//   int j;
//   double dist = 0;
//   for (j=0; j<D; j++)
//     dist += (y[first][j] - x[second][j]) * (y[first][j] - x[second][j]);
//   return dist;
// }
