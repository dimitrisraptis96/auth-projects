#include "hip/hip_runtime.h"
#include "../include/parallel.h"
#include "../include/global_vars.h"
#include "../include/cuda_helpers.h"

// constant device N and D
__device__ __constant__ int N_SIZE; 
__device__ __constant__ int D_SIZE;

// grid and block sizes
const int threads_per_block = 256;
const int blocks_per_grid = 32; 

typedef struct {
    int j;
    double distance;
} SparseData;

// host copies
double *x_data, *y_data, *buffer, **x, **y;
SparseData **w; 

// device copies
double *d_x_data,*d_y_data,*d_y_new_data,*d_m_data,*d_norm, *d_buffer;
double **d_x,**d_y,**d_y_new,**d_m;
int *d_nNbr;
SparseData **d_w; 

__global__ void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData **w);
__global__ void gpu_normalize(int *nNbr, SparseData **w, double *y_new);

extern "C"
void parallel(){
  printf("[INFO]: CUDA-GPU IMPLEMENTATION\n");
  printf("=============================\n");

  struct timeval startwtime, endwtime;
  double seq_time;

  init_parallel();
  
  gettimeofday (&startwtime, NULL);
  //------------------------------
  meanshift();
  //------------------------------
  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);

  /*printf("\n\nIs test PASSed? %s\n\n", validate_parallel()?"YES":"NO");
  printf("===============================================\n\n");*/
  printf("\n\n[INFO]: Parallel meanshift wall clock time = %f\n", seq_time);

}

void init_parallel(){
  // blocks_per_grid = MIN(32, (N+threads_per_block-1) / threads_per_block);

  cpu_malloc();
  gpu_malloc();
  read_file();
  move_data_to_gpu();
}

//Contiguous memory allocation for 2D arrays
void cpu_malloc(){
  int i;
  
  if(VERBOSE) printf("[INFO]: Allocating cpu memory..\n");

  // malloc pointers to rows 
  HANDLE_NULL( (x = (double **) malloc(N * sizeof(double *))) );
  HANDLE_NULL( (y = (double **) malloc(N * sizeof(double *))) );
  HANDLE_NULL( (w = (SparseData **) malloc(N * sizeof(SparseData *))) );

  // malloc data of the arrays
  HANDLE_NULL( (x_data = (double *) malloc(N * D * sizeof(double))) );
  HANDLE_NULL( (y_data = (double *) malloc(N * D * sizeof(double))) );

  // assign pointers of data to arrays
  for(i=0; i < N; i++){
    x[i]      = x_data      + i * D;
    y[i]      = y_data      + i * D;
  }
}


// Allocate memory for devive variables and copy x host data to device
void gpu_malloc (){
  int size; 

  if(VERBOSE) printf("[INFO]: Allocating device memory..\n");
  
  // malloc pointers of rows
  size = N * sizeof(double *);
  HANDLE_ERROR( hipMalloc((void**)&d_x,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m,    size) );


  // malloc data of the arrays
  size = N * D * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_x_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_data,    size) );
  HANDLE_ERROR( hipMalloc((void**)&d_y_new_data,size) );
  HANDLE_ERROR( hipMalloc((void**)&d_m_data,    size) );

  // malloc d_norm
  size = threads_per_block * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_norm, size) );

  // malloc d_nNbr
  size = N * sizeof(int);
  HANDLE_ERROR( hipMalloc((void**)&d_nNbr, size) );

  // malloc d_buffer
  size = N * sizeof(double);
  HANDLE_ERROR( hipMalloc((void**)&d_buffer, size) );

  // malloc d_w indexes of rows
  size = N * sizeof(SparseData *);
  HANDLE_ERROR( hipMalloc((void**)&d_w, size) );
}

void move_data_to_gpu(){
  if(VERBOSE) printf("[INFO]: Move data to device..\n");

  // move device constant variables N_SIZE and D_SIZE
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(N_SIZE), &N, sizeof(int)) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE), &D, sizeof(int)) );

  // move device global variable d_x and d_x_data
  HANDLE_ERROR( hipMemcpy(d_x,      x,      N*sizeof(double *), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_x_data, x_data, N*D*sizeof(double), hipMemcpyHostToDevice) );
}

void free_memory(){
  int i;
  if (VERBOSE) printf("[INFO]: Deallocating memory...\n");
  //free() data
  for (i=0; i<N; i++){
    free(x[i]);
    free(y[i]);
  }
  // free() pointers
  free(x);
  free(y);
}

void read_file(){
  int i,j;

  FILE * fp;
  fp = fopen (DATASET_PATH, "r");

  if (fp == NULL) { perror("[ERROR]: "); exit(1); }

  for (i=0; i<N; i++) 
    for (j=0; j<D; j++)
      if (EOF ==  fscanf(fp, "%lf", &x[i][j])) { perror("[ERROR]:"); exit(1); }

  fclose(fp);
}

void write_csv_file (char *message, double **a, const int ROW, const int COL){
  int i,j;

  FILE * fp;
  fp = fopen (OUTPUT_PATH, "w");

  if (fp == NULL){ perror("[ERROR]: "); exit(1); }

  fprintf(fp,"%s",message);

  for (i=0; i<ROW; i++) {
    for (j=0; j<COL; j++)
      if (EOF ==  fprintf(fp, "%lf, ", a[i][j])) {
        perror("[ERROR]:"); exit(1);
      }
    fprintf(fp,"\n");
  }

  fclose(fp);
}


void meanshift(){
  int iter=0;
  double norm = DBL_MAX;

  gpu_init_arr <<<blocks_per_grid, threads_per_block>>> (d_nNbr, d_x_data, d_y_data, d_m_data);

  while (norm > EPSILON){
    iter++;
    printf("-1\n");
    // find distances and calculate kernels
    rangesearch2sparse();

    // compute new y vector
    gpu_matrix_mult <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_x_data,d_y_data,d_w);

    // normalize vector
    gpu_normalize <<<blocks_per_grid, threads_per_block>>>(d_nNbr,d_w,d_y_new_data);    

    // calculate meanshift
    gpu_calc_meanshift <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_y_new_data,d_y_data);

    // update y
    gpu_copy_2Darray <<<blocks_per_grid, threads_per_block>>>(d_y_new_data, d_y_data);

    // calculate Frobenius norm
    gpu_frob_norm <<<blocks_per_grid, threads_per_block>>>(d_m_data,d_norm);
    
    //wait kernel calls to be executed
    HANDLE_ERROR( hipDeviceSynchronize() );

    norm = finish_norm();

    if (VERBOSE){
      printf("[INFO]: Iteration %d - error %lf\n", iter, norm);
    }
  }

  // copy results back to host
  if (VERBOSE){  
    HANDLE_ERROR( hipMemcpy(y_data, d_y_data, N*D*sizeof(double), hipMemcpyDeviceToHost) );
    write_csv_file("",y,N,D);
  }

  // gpu_free_memory();
  // cpu_free_memory();
}

// TODO: shared memory: the data within the block
__global__
void gpu_init_arr(int *nNbr, double *x, double *y, double *m)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;

  while (tid < N_SIZE*D_SIZE) {
    nNbr[tid%N_SIZE] = 0;
    y[tid] = x[tid];
    m[tid] = DBL_MAX;
    
    tid += blockDim.x * gridDim.x;
  }
}

// TODO: reduction with shared memory
__global__ void gpu_find_neighbours(int y_row, double h, int *nNbr, double *buffer, double *y, double *x){
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int i,j, x_offset, y_offset;
  double dist;

  i = tid/D_SIZE;
  j = tid%D_SIZE;

  while (tid < N_SIZE*D_SIZE) {
    // diagonal elements
    if (y_row == i){
      buffer[i] = 1; nNbr[i]++;
      return;
    }
    
    x_offset = i*D_SIZE;
    y_offset = y_row*D_SIZE;
    
    dist += (y[j + y_offset] - x[j + x_offset])*(y[j + y_offset] - x[j + x_offset]);

    if (dist < h*h){
        buffer[i]= exp(-dist / (2.0*h*h));
        nNbr[i]++;
    }
    // unnecessary points
    else{
      buffer[i]=0;
    }

    tid += blockDim.x * gridDim.x;
  }
}

void rangesearch2sparse(){
  int i,j, index, size;
  double *buffer;

  // malloc buffer for sparse matrix's rows
  HANDLE_NULL( (buffer = (double *) malloc(N*sizeof(double))) );

  for (i=0; i<N; i++){
    // find neighbours of y[i] row
    gpu_find_neighbours <<<blocks_per_grid, threads_per_block>>>(i,BANDWIDTH,d_nNbr,d_buffer,d_y_data,d_x_data);
    HANDLE_ERROR( hipMemcpy(buffer, d_buffer, N*sizeof(double), hipMemcpyDeviceToHost) );

    // get number of neighbours for y[i] from device
    // HERE MAYBE THERE IS AN ERROR
    HANDLE_ERROR( hipMemcpy(&size, &d_nNbr[i], sizeof(int), hipMemcpyDeviceToHost) );
    // malloc w[i] device memory
    HANDLE_ERROR( hipMalloc((void**)&d_w[i], size * sizeof(SparseData)) );

    index = 0;
    SparseData tmp;
    for (j=0; j<N; j++){
      if (buffer[j] > 0){
        tmp.j        = j;
        tmp.distance = buffer[j];
        // HERE MAYBE THERE IS AN ERROR
        HANDLE_ERROR( hipMemcpy(&d_w[i][index], &tmp, sizeof(SparseData), hipMemcpyHostToDevice) );
        index++;
      }
    }
  }
  free(buffer);
}


__global__
void gpu_matrix_mult(int *nNbr, double *x, double *y, SparseData **w)
{
  int tid = threadIdx.x  + blockIdx.x*blockDim.x;
  int k, offset, i, j;
  
  i = tid/D_SIZE;
  j = tid%D_SIZE;

  while(tid < N_SIZE*D_SIZE){
    offset = i*D_SIZE;
    y[tid] = 0;
    for(k=0; k<nNbr[i]; k++)
      y[tid] += w[i][k].distance * x[ w[i][k].j + offset ];  
    tid += blockDim.x*gridDim.x;
  }
}

// TODO: reduction using shared memory
__global__ void gpu_normalize(int *nNbr, SparseData **w, double *y_new) 
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,k;
  double sum=0;
  
  i = tid/D_SIZE;
  j = tid%D_SIZE;

  while(tid < N_SIZE*D_SIZE) {
    for (int k=0; k<nNbr[i]; k++)
      sum += w[i][j].distance; // not contiguous memory, [][] necessary

    y_new[tid] /= sum;
    tid += gridDim.x*blockDim.x;
  }
}

__global__ void gpu_calc_meanshift(double *m, double *y_new, double *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    m[tid] = y_new[tid] - y[tid];
    tid += gridDim.x+blockDim.x;
  }
}

__global__ void gpu_copy_2Darray(double *source, double *destination)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  while(tid < N_SIZE*D_SIZE){
    destination[tid] = source[tid];
    tid += gridDim.x+blockDim.x;
  }
}

// TODO: non-shared implementation (use code from gpu_normalize)
__global__ void gpu_frob_norm(double *m, double *result){
  __shared__ float cache[threads_per_block];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  double norm = 0;
  while (tid < N_SIZE*D_SIZE) {
      norm += m[tid] * m[tid];
      tid += blockDim.x*gridDim.x;
  }
  
  // set the cache values
  cache[cacheIndex] = norm;
  
  // synchronize threads in this block
  __syncthreads();

  // for reductions, threads_per_block must be a power of 2
  int i = blockDim.x/2;
  while (i != 0) {
      if (cacheIndex < i)
          cache[cacheIndex] += cache[cacheIndex + i];
      __syncthreads();
      i /= 2;
  }

  if (cacheIndex == 0)
      result[blockIdx.x] = cache[0];
}

// calculate last step of norm on CPU because it's more efficient
double finish_norm(){
  double *result, norm;

  // malloc result array
  HANDLE_NULL( (result = (double *) malloc(threads_per_block*sizeof(double))) );

  HANDLE_ERROR( hipMemcpy( result, 
                            d_norm,
                            blocks_per_grid*sizeof(float),
                            hipMemcpyDeviceToHost ) );
  norm = 0;
  for (int i=0; i<blocks_per_grid; i++)
      norm += result[i];
  return sqrt(norm);
}

void print_2Darray(double **a, const int ROW, const int COL){
  int i,j;
  for (i=0;i<ROW;i++){
    for (j=0; j<COL; j++){
      printf("%lf \t",a[i][j]);
    }
  printf("\n");
  }
}
